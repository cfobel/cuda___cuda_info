#include <hip/hip_runtime.h>

int main(int argc, char** argv) {
	int major = 0;
    int minor = 0;
	int deviceCount = 0;

	hipError_t err = hipInit(0);
    hipGetDeviceCount(&deviceCount);

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
        return 0;
	}

    // This function call returns 9999 for both major & minor fields, 
    // if no CUDA capable devices are present
    hipDeviceComputeCapability(&major, &minor, 0);
    while(1);

    return 0;
}
